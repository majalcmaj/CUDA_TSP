#include "hip/hip_runtime.h"
#include"cudatsp.h"
#include"tspcommon.h"
#include <stdio.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#include<>
#include<hiprand.h>

#include<time.h>

#define BLOCKS_COUNT 16

// DEBUG
#include"../DataIO/dataio.h"

__global__ void
calcDistanceForEachPermutation(location_t* locations, int* idx_holder, dist_idx_t* distances);

__global__ void
populateMemory(int* idx_holder, int locations_count, double* rands);

__global__ void
rankPermutations(dist_idx_t* distances);

int run_cuda_tsp(location_t* locations, int locations_count)
{
	double* rands;
	int* idx_holder;
	dim3 gridDim(BLOCKS_COUNT);
	dim3 blockDim(locations_count);
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
	CudaCheckError();
	CudaSafeCall(hipMalloc(&rands, sizeof(double) * locations_count * BLOCKS_COUNT));
	hiprandGenerateUniformDouble(gen, rands, locations_count * BLOCKS_COUNT);
	CudaCheckError();

	CudaSafeCall(hipMalloc(&idx_holder, sizeof(int) * BLOCKS_COUNT * locations_count));
	populateMemory<<<gridDim, blockDim>>>(idx_holder, locations_count, rands);
	CudaCheckError();

	int* idx = (int*)malloc(sizeof(int) * locations_count * BLOCKS_COUNT);

	location_t* locd;
	int locsize = sizeof(location_t) * locations_count;
	hipMalloc(&locd, locsize);
	hipMemcpy(locd, locations, locsize, hipMemcpyHostToDevice);

	dist_idx_t* distances = (dist_idx_t*)malloc(BLOCKS_COUNT * sizeof(dist_idx_t));
	dist_idx_t* distances_d;
	hipMalloc(&distances_d, 2 * BLOCKS_COUNT * sizeof(dist_idx_t));
	calcDistanceForEachPermutation<<<gridDim, blockDim, sizeof(double) * locations_count>>>(locd, idx_holder, distances_d);
	CudaCheckError();

	rankPermutations<<<1, BLOCKS_COUNT, 2 * sizeof(dist_idx_t) * BLOCKS_COUNT>>>(distances_d);
	CudaCheckError();
	// DEBUG
	hipMemcpy(distances, distances_d, BLOCKS_COUNT * sizeof(dist_idx_t), hipMemcpyDeviceToHost);
	hipMemcpy(idx, idx_holder, sizeof(int) * BLOCKS_COUNT * locations_count, hipMemcpyDeviceToHost);
	for(int i = 0 ; i < BLOCKS_COUNT; i++)
	{
		printf("Distance for block %lf with index %d\n", distances[i].distance, distances[i].index);
//		for(int j = 0 ; j < locations_count ; j++)
//		{
//			printf("%d ", idx[i * locations_count + j]);
//		}
//		printf("\n");
	}
	return 0;
}
