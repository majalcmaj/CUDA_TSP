
#include <stdio.h>
#include"common.h"
#include "DataIO/dataio.h"
#include "CudaTSP/cudatsp.h"

int main()
{
	int errno = 0;
	location_t* locations = NULL;
	int locations_length;

	errno = read_csv("../resources/CitiesLocationsSmall.csv", &locations, &locations_length);
	if(errno != 0)
	{
		fprintf(stderr, "Reading city locations from file failed. with errno=%d\n", errno);
		return 1;
	}

	run_cuda_tsp(locations, locations_length);

	free(locations);
	return 0;
}

//// System includes
//#include <stdio.h>
//// CUDA runtime
//#include <hip/hip_runtime.h>
//#include<>
//
//int main()
//{
//	int count;
//	hipGetDeviceCount(&count);
//
//	printf("Available devices: %d\n", count);
//	hipDeviceProp_t prop;
//
//	for(int i = 0; i < count; i++)
//	{
//		hipGetDeviceProperties(&prop, i);
//		printf("Device: %d: %s\n", i, prop.name);
//		printf("Compute capability: %d.%d\n", prop.major, prop.minor);
//		printf("Max grid dims: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
//		printf("Max block dims: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
//		printf("Shared mem per block %d\n", prop.sharedMemPerBlock);
//		printf("Max threads per block %d\n", prop.maxThreadsPerBlock);
//	}
//	return 0;
//}
