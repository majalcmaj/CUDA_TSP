/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>


__global__ void addArrays(int* a, int* b, int* c )
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int add_arrays()
{
	const int count = 5;
	const int size = count * sizeof(int);
	int ha[] = { 1, 2, 3, 4, 5 };
	int hb[] = { 10, 20, 30, 40, 50 };
	int hc[count];

	int *da, *db, *dc;
	hipMalloc(&da, size);
	hipMalloc(&db, size);
	hipMalloc(&dc, size);

	hipMemcpy(da, ha, size, hipMemcpyHostToDevice);
	hipMemcpy(db, hb, size, hipMemcpyHostToDevice);

	addArrays<<<1, count>>>(da, db, dc);

	hipMemcpy(hc, dc, size, hipMemcpyDeviceToHost);

	for(int i = 0 ; i < count ; i++) {
		printf("%d ", hc[i]);
	}
}
