// System includes
#include <stdio.h>
#include <hip/hip_runtime.h>

#include<hiprand/hiprand.h>
#define _USE_MATH_DEFINES
#include<math.h>


__global__ void sumSingleBlock(int *d)
{
	int tid = threadIdx.x;
	int myIdx = tid * 2;
	int diff = 1;
	while(myIdx + diff < 2 * blockDim.x) {
		d[myIdx] += d[myIdx + diff];
		diff <<= 1;
		__syncthreads();
	}
}
int main()
{
	const int count = 512;
	const int size = count * sizeof(int);

	int h[count];
	for(int i = 0 ; i < count; i++) {
		h[i] = 1 + i;
	}
	int *d;
	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);
	sumSingleBlock<<<1, count/2>>>(d);

	int result;
	hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);

	printf("Result: %d\n", result);
//	cudaMemcpy(h, d, size, cudaMemcpyDeviceToHost);
//	for(int i = 0 ; i < count ; i++) {
//		printf("element %d: %d\n", i, h[i]);
//	}
	hipFree(d);
	return 0;
}
