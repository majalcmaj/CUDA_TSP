#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void scan(int *d)
{
	int myIdx = threadIdx.x;
	int diff=1;
	while(myIdx + diff < blockDim.x + 1)
	{
		d[myIdx + diff] += d[myIdx];
		diff <<= 1;
		__syncthreads();
	}
}

int main()
{
	const int count = 16;
	const int size = count * sizeof(int);

	int h[16];
	for(int i = 0; i < count ; i++) {
		h[i] = i + 1;
	}
	int *d;
	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	scan<<<1, count-1>>>(d);

	hipMemcpy(h, d, size, hipMemcpyDeviceToHost);
	for(int i = 0 ; i < count ; i++) {
		printf("%d: %d\n", i, h[i]);
	}
}
