#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void scan(int *d)
{
	extern __shared__ int dcopy[];
	int myIdx = threadIdx.x;
	int diff=1;
	dcopy[myIdx] = d[myIdx];
	while(myIdx + diff < blockDim.x)
	{
		dcopy[myIdx + diff] += dcopy[myIdx];
		diff <<= 1;
		__syncthreads();
	}
	d[myIdx] = dcopy[myIdx];
}

int main()
{
	const int count = 16;
	const int size = count * sizeof(int);

	int h[16];
	for(int i = 0; i < count ; i++) {
		h[i] = i + 1;
	}
	int *d;
	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	scan<<<1, count, size>>>(d);

	hipMemcpy(h, d, size, hipMemcpyDeviceToHost);
	for(int i = 0 ; i < count ; i++) {
		printf("%d: %d\n", i, h[i]);
	}
}
