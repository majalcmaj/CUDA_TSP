// System includes
#include <stdio.h>
#include<time.h>
// CUDA runtime
#include <hip/hip_runtime.h>

#include<hiprand/hiprand.h>

__global__ void addTen(float* d, int count)
{
	int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
	int threadPosInBlock = threadIdx.x +
			blockDim.x * threadIdx.y +
			blockDim.x * blockDim.y * threadIdx.z;
	int blockPosInGrid = blockIdx.x +
			gridDim.x * blockIdx.y +
			gridDim.x * gridDim.y * blockIdx.z;
	int tid = blockPosInGrid * threadsPerBlock + threadPosInBlock;
	if(tid < count) {
		d[tid] += 10;
	}
}

int simple_map()
{
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(gen, time(0));

	const int count = 123456;
	const int size = count * sizeof(float);
	float *d;
	float h[count];
	hipMalloc(&d, size);

	hiprandGenerateUniform(gen, d, count);

	dim3 block(8, 8, 8);
	dim3 grid(16, 16);

	addTen<<<grid, block>>>(d, count);
	hipMemcpy(h, d, size, hipMemcpyDeviceToHost);
	for(int i = 0 ; i < 100 ; i++) {
		printf("%f \n", h[i]);
	}
	return 0;
}
